
#include <hip/hip_runtime.h>
/**
 * @file matrix_mult_dot_naive_double.cu
 *
 * CUDA code to calculate D = (A*B).*C using a naive algorithm (double precision)
 *
 */


/** Main entry point.
 * Implements naive double precision matrix multiplication and element wise multiplication
 */
__global__ void matrix_mult_dot_naive_double(
                      double * D,
                      const double * A,
                      const double * B,
		      const double * C,
                      const unsigned int A_B_sum_length,
                      const unsigned int columns_D,
                      const unsigned int rows_D) {

    // Work out which thread we are
    int row_D_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int column_D_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform matrix multiplication
    if (row_D_idx < rows_D && column_D_idx < columns_D){
        double tmp_sum = 0.0;
        for (int k_idx = 0; k_idx < A_B_sum_length; k_idx++){
            tmp_sum += A[row_D_idx * A_B_sum_length + k_idx] * B[k_idx * columns_D + column_D_idx];
        }
        // Place in the output array
        D[row_D_idx * columns_D + column_D_idx] = tmp_sum * C[row_D_idx * columns_D + column_D_idx];
    }
}

